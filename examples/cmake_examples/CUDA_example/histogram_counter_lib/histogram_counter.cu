#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include "jpeg_reader.hpp"
#include "histogram_counter.cuh"
#include <hip/hip_runtime.h>

// function to add the elements of two arrays
// CUDA Kernel function to add the elements of two arrays on the GPU
// __global__ functions are "kernels", running on device. 
__global__ void cuda_add(int n, float *x, float *y)
{
  // This is the same operation done on each thread.
  // for (int i = 0; i < n; i++)
  //     y[i] = x[i] + y[i];
  // stride thru its index
  unsigned int stride = blockDim.x; 
  for(unsigned int i = threadIdx.x; i < n; i += stride){
      y[i] = x[i] + y[i];
  }

}

void add(int N, float x_val, float y_val){
  // float *x = new float[N];
  // float *y = new float[N];
  float *x, *y;
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = x_val;
    y[i] = y_val;
  }

  // Run kernel on 1M elements on the CPU
  cuda_add<<<1,8>>>(N, x, y);   
  //doing the same computation once per thread, <<<1, num_threads>>>
  // num_threads should be a power of 2, and it will be blockDim.x. if not power of 2, no errors, but not as efficient

  // Wait for GPU to finish before accessing on host, cuz CUDA doesn't block CPU threads
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  // delete [] x;
  // delete [] y;
  hipFree(x);
  hipFree(y);
}

